
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <cstdio>
#include <cmath>

int main() {
  // Init in cpu
  float *inp_data_cpu = (float*)malloc(sizeof(float) * 2);
  for(int i=0; i<2; i++) {
    inp_data_cpu[i] = i;
  }

  // Copy to device and allocate cuda memory and set data
  hipfftComplex *data;
  hipMalloc((void**)&data, sizeof(hipfftComplex) * 1);
  hipMemcpy(data, inp_data_cpu, sizeof(hipfftComplex) * 1, hipMemcpyHostToDevice);

  // Copy back to cpu
  float *out_data_cpu = (float*)malloc(sizeof(float) * 2);
  hipMemcpy(out_data_cpu, data, sizeof(hipfftComplex) * 1, hipMemcpyDeviceToHost);
  for(int i=0; i<2; i++) {
    printf("out_data_cpu[%d] = %f\n", i, out_data_cpu[i]);
  }
}
